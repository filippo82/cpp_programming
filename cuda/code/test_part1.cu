#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <time.h>
#include <stdio.h>

// CPU version - Serial - One thread
void add_cpu(int n, float *x, float *y)
{
   for (int i=0; i<n; i++)
   {
      y[i] = cos(x[i]) + sin(y[i]);
   }
}

// Kernel function to add the elements of two arrays
__global__
void add_gpu(int n, float *x, float *y, int a)
{
  // int index = threadIdx.x;
  // int stride = blockDim.x;
  // int check = 0;
  // int index = blockIdx.x * blockDim.x + threadIdx.x;
  // int stride = blockDim.x * gridDim.x;
  // if (check == 0) {
  //     printf("Stride for blockDim = %d is %d\n", a, stride);
  //     check = 1;
  // }
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
  {
    // y[i] = x[i] + y[i];
    y[i] = a;
    // y[i] = cos(x[i]) + sin(y[i]);
  }
}

void checkCudaDevices()
{
  int nDevices;
  hipGetDeviceCount(&nDevices);
  for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
      printf("  Major: %d\n", prop.major);
      printf("  Minor: %d\n", prop.minor);
      printf("  Multiprocess count: %d\n", prop.multiProcessorCount);
      printf("  Max dimension size of a grid size (x,y,z): (%d, %d, %d)", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
      // printf("  Max grid size: %d\n", prop.maxGridSize[0]);
      printf("\n");
  }
}

int main(void)
{
  checkCudaDevices();
  int numSMs;
  int devId = 0;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, devId);
  // std::cout << "NumSMs: " << numSMs << std::endl;

  int n = 1 << 20; // Around 4GB
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, n*sizeof(float));
  hipMallocManaged(&y, n*sizeof(float));

  // Initialize x and y arrays on the host
  for (int i = 0; i < n; i++)
  {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  double avg = 0;
  clock_t t;

  // Runs add 10 times on CPU
  // for(int i=0; i<10; i++)
  // {
  //   t = clock(); //start time
  //   add_cpu(n, x, y);
  //   t = clock() - t; //total time = end time - start time
  //   printf("CPU RUN-%d time = %f ms.\n",i,(((float)t)/CLOCKS_PER_SEC)*1000);
  //   avg += ((((float)t)/CLOCKS_PER_SEC)*1000);//time is calculated in terms of clockcycle. Converted in millisecond
  // }
  // std::cout << "The average time on CPU is: " << avg / 10.0 << " ms" << std::endl;

  // Run kernel on 20M elements on the GPU
  // add_gpu<<<1, 512>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  // hipDeviceSynchronize();

  int data[10];

  // Runs add 10 times on GPU
  for(int i=32; i<=1024; i+=32)
  {
    avg = 0;
    for(int j=0; j<1; j++)
    {
      t = clock(); // Start time
      int blockSize = 1024 - i + 32; // Threads per block
      int numBlocks = (n + blockSize - 1) / blockSize;
      // add_gpu<<<dim3(i, 1, 1), dim3(16, 1, 1)>>>(n, x, y);
      add_gpu<<<numBlocks, blockSize>>>(n, x, y, blockSize);
      // add_gpu<<<256*numSMs, blockSize>>>(n, x, y);
      hipDeviceSynchronize();
      t = clock() - t; // Total time = end time - start time
      printf("<<<%d, %d>>> - GPU RUN-%d time = %f ms.\n",numBlocks, blockSize, j,(((float)t)/CLOCKS_PER_SEC)*1000);
      avg += ((((float)t)/CLOCKS_PER_SEC)*1000);// Time is calculated in terms of clockcycle. Converted in millisecond
    }
    data[i] = avg / 10.0;
    // std::cout << "The average time on GPU is: " << data[i] << " ms" << std::endl;
    std::cout << y[100] << std::endl;
  }

  // std::cout << y[1] << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  return 0;
}
